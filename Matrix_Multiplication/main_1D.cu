#include <bits/stdc++.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

void mul_cpu(int row_A, int col_A, int col_B, int* mat_A, int* mat_B, int* mat_C){
    for(int i = 0; i < row_A; i++){
        for(int j = 0; j < col_B; j++){
            for(int k = 0; k < col_A; k++){
                // printf("%d x %d, ", mat_A[i][k], mat_B[k][j]);
                mat_C[i * col_B + j] += mat_A[i * col_A + k] * mat_B[k * col_B + j];
            }
            // printf("\n");
        }
    }
}

__global__ void mul_cuda(int row_A, int col_A, int col_B, int* mat_A_CUDA, int* mat_B_CUDA, int* mat_C_CUDA){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < row_A && col < col_B && row >= 0 && col >= 0){
        for(int k = 0; k < col_A; k++){
            mat_C_CUDA[row * col_B + col] += mat_A_CUDA[row * col_A + k] * mat_B_CUDA[k * col_B + col];
        }        
    }
}


int* init(int row, int col, bool is_C){
    int* mat = (int*) malloc(row * col * sizeof(int *));

    random_device rd;
    mt19937 generator(rd());
    uniform_int_distribution<int> unif(INT_MIN, INT_MAX);

    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){
            mat[i * col + j] = is_C ? 0 : unif(generator);
        }
    }

    return mat;
}

int main(int argc, char* argv[]){
    /*-------------- CPU init ------------*/
    int row_A, col_A, col_B;
    int* mat_A;
    int* mat_B;
    int* mat_C;
    int* mat_A_CUDA;
    int* mat_B_CUDA;
    int* mat_C_CUDA;
    int* res_CPU;
    int* res_GPU;

    if(argc != 5){
        fprintf(stderr, "%s", "Usage: ./a.out $row_A $col_A $col_B BLOCK_SIZE in 1Dim direction\n");
        exit(-1);
    }

    row_A = atoi(argv[1]);
    col_A = atoi(argv[2]);
    col_B = atoi(argv[3]);
    assert(row_A > 0 && col_A > 0 && col_B > 0);

    mat_A = init(row_A, col_A, false);
    mat_B = init(col_A, col_B, false);
    mat_C = init(row_A, col_B, true);
    res_CPU = init(row_A, col_B, true);

    /*-------------- CPU run -------------*/
    struct timeval start, end;
    gettimeofday(&start, 0);
    mul_cpu(row_A, col_A, col_B, mat_A, mat_B, mat_C);
    gettimeofday(&end, 0);
    int sec = end.tv_sec - start.tv_sec;
    int usec = end.tv_usec - start.tv_usec;
    int t_cpu = sec * 1000 + (usec / 1000);
    printf("CPU serial time (ms): %d\n", t_cpu);

    /*------------- Clear ---------------*/
    res_CPU = init(row_A, col_B, true);
    for(int i = 0; i < row_A; i++){
        for(int j = 0; j < col_B; j++){
            res_CPU[i * col_B + j] = mat_C[i * col_B + j];
            mat_C[i * col_B + j] = 0;
        }
    }
    /*-------------- CUDA init ------------*/
    hipError_t ce_A, ce_B, ce_C;

    ce_A = hipMalloc((void**) &mat_A_CUDA, row_A * col_A * sizeof(int));
    ce_B = hipMalloc((void**) &mat_B_CUDA, col_A * col_B * sizeof(int));
    ce_C = hipMalloc((void**) &mat_C_CUDA, row_A * col_B * sizeof(int));
    if( ce_A != hipSuccess ||
        ce_B != hipSuccess || 
        ce_C != hipSuccess){
        fprintf(stderr, "%s", "hipMalloc failed\n");
        exit(1);
    }

    ce_A = hipMemcpy(mat_A_CUDA, mat_A, row_A * col_A * sizeof(int), hipMemcpyHostToDevice);
    ce_B = hipMemcpy(mat_B_CUDA, mat_B, col_A * col_B * sizeof(int), hipMemcpyHostToDevice);
    ce_C = hipMemcpy(mat_C_CUDA, mat_C, row_A * col_B * sizeof(int), hipMemcpyHostToDevice);
    if( ce_A != hipSuccess ||
        ce_B != hipSuccess || 
        ce_C != hipSuccess){
        fprintf(stderr, "%s", "hipMemcpyHostToDevice failed\n");
        exit(2);
    }

    const int BLOCK_SIZE = (int)sqrt(atoi(argv[4]));
    // configured as method 2 in https://blog.csdn.net/yongjiankuang/article/details/90180559
    const dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 dim_grid((row_A + BLOCK_SIZE + 1) / BLOCK_SIZE,(col_B + BLOCK_SIZE + 1) / BLOCK_SIZE);

    /*-------------- CUDA run -------------*/
    gettimeofday(&start, 0);
    mul_cuda<<<dim_grid, dim_block, 0>>>(row_A, col_A, col_B, mat_A_CUDA, mat_B_CUDA, mat_C_CUDA);
    hipError_t ce_K; // cuda erroe for kernel
    ce_K = hipDeviceSynchronize();
    if(ce_K != hipSuccess){
        fprintf(stderr, "%s", "hipDeviceSynchronize failed\n");
        exit(3);
    }
    gettimeofday(&end, 0);
    sec = end.tv_sec - start.tv_sec;
    usec = end.tv_usec - start.tv_usec;
    int t_gpu = sec * 1000 + (usec / 1000);
    cout << "GPU CUDA time (ms): " << t_gpu << '\n';

    /*------- Check integrity -------------*/
    res_GPU = init(row_A, col_B, true);
    ce_C = hipMemcpy(res_GPU, mat_C_CUDA, row_A * col_B * sizeof(int), hipMemcpyDeviceToHost);
    if(ce_C != hipSuccess){
        fprintf(stderr, "%s", "hipMemcpyDeviceToHost failed\n");
        exit(4);
    }

    printf("Check integrity\n");
    for(int i = 0; i < row_A; i++){
        for(int j = 0; j < col_B; j++){
            assert(res_CPU[i * col_B + j] == res_GPU[i * col_B + j]);
        }
    }
    printf("Integrity pass!, CPU result == GPU result, all finished\n");
    printf("[row_A, col_A, col_B, Accelerate ratio (times)]: \n");
    printf("%d, %d, %d, %f\n", row_A, col_A, col_B, (float)t_cpu / (float)t_gpu);

    /*------- Clear memory -------------*/
    hipFree(mat_A_CUDA);
    hipFree(mat_B_CUDA);
    hipFree(mat_C_CUDA);
    free(mat_A);
    free(mat_B);
    free(mat_C);

    return 0;
}